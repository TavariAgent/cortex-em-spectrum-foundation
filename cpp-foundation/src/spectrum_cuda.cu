#include "hip/hip_runtime.h"
#include "spectrum_cuda.hpp"

#ifdef CUDA_AVAILABLE
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdint.h>

namespace cortex {

static __device__ __forceinline__ float clamp01f(float v){ return v<0.f?0.f:(v>1.f?1.f:v); }

static __device__ __forceinline__ void base_rgb(float wl, float& r, float& g, float& b) {
    r=g=b=0.f;
    if (wl>=380.f && wl<440.f){ r=-(wl-440.f)/60.f; b=1.f; }
    else if (wl>=440.f && wl<490.f){ g=(wl-440.f)/50.f; b=1.f; }
    else if (wl>=490.f && wl<510.f){ g=1.f; b=-(wl-510.f)/20.f; }
    else if (wl>=510.f && wl<580.f){ r=(wl-510.f)/70.f; g=1.f; }
    else if (wl>=580.f && wl<645.f){ r=1.f; g=-(wl-645.f)/65.f; }
    else if (wl>=645.f && wl<=750.f){ r=1.f; }
    r=clamp01f(r); g=clamp01f(g); b=clamp01f(b);
}

static __device__ __forceinline__ float intensity_term(float wl) {
    float I=1.f;
    if (wl>=380.f && wl<420.f) I = 0.3f + 0.7f*(wl-380.f)/40.f;
    else if (wl>=701.f && wl<=750.f) I = 0.3f + 0.7f*(750.f-wl)/49.f;
    return I;
}

static __device__ __forceinline__ uint32_t xorshift32(uint32_t& s){ s^=s<<13; s^=s>>17; s^=s<<5; return s; }
static __device__ __forceinline__ float rand01(uint32_t& s){ return (xorshift32(s)&0x00FFFFFF)*(1.0f/16777216.0f); }

__global__ void spectrum_kernel(SpectrumCudaParams p, float* out_rgb) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if (x>=p.width || y>=p.height) return;

    const int sppX = max(1, p.spp_x);
    const int sppY = max(1, p.spp_y);
    const int spp  = sppX*sppY;

    float acc_r=0.f, acc_g=0.f, acc_b=0.f;
    uint32_t seed = (uint32_t)((y*p.width+x)*9781u + 0x9E3779B9u);

    for (int sy=0; sy<sppY; ++sy){
        for (int sx=0; sx<sppX; ++sx){
            float jx = p.jitter ? rand01(seed) : 0.5f;
            float fx = (sx + jx) / (float)sppX;
            float xN = (x + fx) / (float)p.width;
            float wl = (float)p.wl_min + ((float)p.wl_max - (float)p.wl_min) * xN;

            float r,g,b;
            base_rgb(wl, r,g,b);
            float I = intensity_term(wl);
            float inv_gamma = (p.gamma>0.0) ? (1.0f/(float)p.gamma) : (1.0f/2.2f);
            r = powf(r*I, inv_gamma);
            g = powf(g*I, inv_gamma);
            b = powf(b*I, inv_gamma);

            acc_r += r; acc_g += g; acc_b += b;
        }
    }

    float scale = 1.f/(float)spp;
    int idx = (y*p.width + x)*3;
    out_rgb[idx+0] = acc_r*scale;
    out_rgb[idx+1] = acc_g*scale;
    out_rgb[idx+2] = acc_b*scale;
}

bool spectrum_shade_cuda(const SpectrumCudaParams& p, float* out_rgb){
    if (!out_rgb || p.width<=0 || p.height<=0) return false;
    float* d_out=nullptr;
    size_t bytes = (size_t)p.width*(size_t)p.height*3*sizeof(float);
    if (hipMalloc(&d_out, bytes)!=hipSuccess) return false;

    dim3 block(16,16);
    dim3 grid((p.width+block.x-1)/block.x, (p.height+block.y-1)/block.y);
    spectrum_kernel<<<grid,block>>>(p, d_out);
    hipError_t e = hipDeviceSynchronize();

    bool ok = (e==hipSuccess);
    if (ok) ok = (hipMemcpy(out_rgb, d_out, bytes, hipMemcpyDeviceToHost)==hipSuccess);
    hipFree(d_out);
    return ok;
}

} // namespace cortex
#else
namespace cortex {
bool spectrum_shade_cuda(const SpectrumCudaParams&, float*) { return false; }
}
#endif